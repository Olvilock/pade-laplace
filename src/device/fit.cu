#include "hip/hip_runtime.h"
#include "polyroot.cuh"
#include "residue.cuh"
#include "taylor_slae.cuh"

#include <pl/fit.cuh>

#include <cuda/std/complex>
#include <cuda/std/span>
#include <cuda/std/tuple>

namespace pl {
using complex = cuda::std::complex<double>;

namespace {
__device__ void
fitBatch(const complex point, BatchResult *out_ptr,
         const cuda::std::pair<complex, complex> transform_pair) {
  extern __shared__ complex roots[];
  auto coeff = roots + blockDim.x;
  auto taylor = coeff + blockDim.x;

  taylor[threadIdx.x] = transform_pair.first;
  taylor[threadIdx.x + blockDim.x] = transform_pair.second;

  // printf("%i: %le %le\n", threadIdx.x,
  //	taylor[threadIdx.x].real(), taylor[threadIdx.x].imag());

  for (int lc_dim = 1; lc_dim <= blockDim.x; out_ptr += lc_dim++) {
    BatchStatus status = BatchStatus::ok;
    __syncthreads();

    if (!(slaeCG(lc_dim, 1000) < 1e-6))
      status |= BatchStatus::degenerate_system;

    auto this_coeff = taylor[threadIdx.x];
    if (threadIdx.x < lc_dim && threadIdx.x) {
      int id_sum = threadIdx.x - 1;
      int id = id_sum / 2;

      if (id_sum % 2)
        this_coeff += taylor[id_sum - id] * coeff[id];
      this_coeff += taylor[id] * coeff[id_sum - id];

      while (id--)
        this_coeff +=
            taylor[id] * coeff[id_sum - id] + taylor[id_sum - id] * coeff[id];
    }

    auto highest_coeff = coeff[threadIdx.x];

    if (!(solveAberth(lc_dim, 100) < 1e-6))
      status |= BatchStatus::Aberth_divergence;

    if (threadIdx.x == lc_dim - 1)
      coeff[0] = highest_coeff;
    __syncthreads();
    for (int id = 1; id < lc_dim; id <<= 1) {
      if (threadIdx.x < id && threadIdx.x + id < lc_dim)
        coeff[threadIdx.x + id] = coeff[threadIdx.x];
      __syncthreads();
    }
    highest_coeff = coeff[threadIdx.x];

    coeff[threadIdx.x] = this_coeff;
    __syncthreads();

    if (threadIdx.x < lc_dim)
      out_ptr[threadIdx.x] = {
          {residue(lc_dim) / highest_coeff, roots[threadIdx.x] + point},
          status};
  }
}
} // namespace

template __global__ void fitTransform(const complex *grid,
                                      BatchResult *result_grid,
                                      cuda::std::span<const DeviceNode>);
template __global__ void
fitTransform(const complex *grid, BatchResult *result_grid,
             cuda::std::span<const SplineSegment> segments, SplineEndpoint left,
             SplineEndpoint right);

template <typename... TransformArgs>
__global__ void fitTransform(const complex *grid, BatchResult *result_grid,
                             TransformArgs... args) {
  const auto point = grid[blockIdx.x];
  fitBatch(point, result_grid + blockIdx.x * blockDim.x * (blockDim.x + 1) / 2,
           transform(point, args...));
}
} // namespace pl
