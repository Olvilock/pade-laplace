#include "hip/hip_runtime.h"
#include <pl/fit.cuh>
#include <pl/spline.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

// error checking macro
#define cudaCheckErrors(msg)                                                   \
  do {                                                                         \
    hipError_t __err = hipGetLastError();                                    \
    if (__err != hipSuccess) {                                                \
      fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", msg,                  \
              hipGetErrorString(__err), __FILE__, __LINE__);                  \
      fprintf(stderr, "*** FAILED - ABORTING\n");                              \
      exit(1);                                                                 \
    }                                                                          \
  } while (false)

namespace {
inline std::ostream& operator <<(std::ostream& out, cuda::std::complex<double> num) {
  return out << (std::complex<double>)num;
}
}

namespace pl {
using complex = cuda::std::complex<double>;
int operator&(BatchStatus a, BatchStatus b) {
  return static_cast<int>(a) & static_cast<int>(b);
}

template <>
[[nodiscard]] Multiexp fit<Method::Trapezia>(const dataset_type &h_data,
                                             unsigned depth) {
  thrust::device_vector<DeviceNode> d_data = h_data;
  auto less_point = [] __device__(const DeviceNode &a, const DeviceNode &b) {
    return a.point < b.point;
  };
  thrust::sort(thrust::device, d_data.begin(), d_data.end(), less_point);

  std::vector<complex> h_grid{
      {5.0, 0.0}, {3.0, 0.0},
      //{ 3.0, 10.0 },
  };
  thrust::device_vector<complex> d_grid = h_grid;
  thrust::device_vector<BatchResult> d_result(
      d_grid.size() * (depth * (depth + 1) / 2), {{}, BatchStatus::untouched});

  hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
  cudaCheckErrors("Device error\n");
  std::cout << "Kernel launch...\n";

  fitTransform<<<d_grid.size(), depth, 4 * depth * sizeof(complex)>>>(
      d_grid.data().get(), d_result.data().get(),
      cuda::std::span<const DeviceNode>(thrust::raw_pointer_cast(d_data.data()),
                                        d_data.size()));

  cudaCheckErrors("Kernel launch failed\n");
  hipDeviceSynchronize();
  cudaCheckErrors("Could not synchronize\n");
  std::cout << "Kernel finished\n";

  thrust::host_vector<BatchResult> h_result = d_result;
  auto res_it = h_result.begin();
  for (auto s : h_grid) {
    std::cout << "For point p = " << s << " we have:\n";
    for (int count = 1; count <= depth; res_it += count++) {
      std::cout << "count = " << count << ":\n";
      auto cur_it = res_it;
      for (int id = 0; id < count; id++, cur_it++) {
        if ((*cur_it).status & BatchStatus::degenerate_system)
          std::cout << "(degenerate)";
        if ((*cur_it).status & BatchStatus::Aberth_divergence)
          std::cout << "(divergence)";
        if ((*cur_it).status & BatchStatus::untouched)
          std::cout << "(untouched)";
        if ((*cur_it).status == BatchStatus::ok)
          std::cout << "(ok)";

        auto data = (*cur_it).data;
        std::cout << "  a_" << id << " = " << data.coeff << ", b_" << id
                  << " = " << data.exp << "\n";
      }
    }
  }
  return {};
}
} // namespace pl
